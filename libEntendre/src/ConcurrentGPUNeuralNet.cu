#include "hip/hip_runtime.h"
#include "ConcurrentGPUNeuralNet.hh"

#include <cassert>
#include <stdexcept>
#include <cmath>
#include <iostream>
#include <map>
#include <set>
#include <sstream>
#include <vector>
#include <algorithm>

#include "logging.h"
#include "math.h"


#define cuda_assert(ans) { cudaAssert((ans), __FILE__, __LINE__); }

inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
//#ifndef NDEBUG
  if (code != hipSuccess) {
    fprintf(stderr,"cuda_assert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
//#endif
}

ConcurrentGPUNeuralNet::~ConcurrentGPUNeuralNet() {
  if (node_) { cuda_assert(hipFree(node_)); }
  if (origin_) { cuda_assert(hipFree(origin_)); }
  if (dest_) { cuda_assert(hipFree(dest_)); }
  if (weight_) { cuda_assert(hipFree(weight_)); }
  if (action_list_) { cuda_assert(hipFree(action_list_)); }
}

ConcurrentGPUNeuralNet::EvaluationOrder ConcurrentGPUNeuralNet::compare_connections(const Connection& a, const Connection& b) {
  // A recurrent connection must be used before the origin is overwritten.
  if (a.type == ConnectionType::Recurrent && a.origin == b.dest) { return EvaluationOrder::LessThan; }
  if (b.type == ConnectionType::Recurrent && b.origin == a.dest) { return EvaluationOrder::GreaterThan; }

  // A normal connection must occur after every connection incoming to its origin has completed.
  if (a.type == ConnectionType::Normal && a.dest == b.origin) { return EvaluationOrder::LessThan; }
  if (b.type == ConnectionType::Normal && b.dest == a.origin) { return EvaluationOrder::GreaterThan; }

  // Two connections writing to the same destination must be in different sets.
  if (a.dest == b.dest) {
    // A self-recurrent connection happens at the same time as
    // zero-ing out, and so must occur first of all connections
    // writing to that node.
    if(a.origin == a.dest) {
      return EvaluationOrder::LessThan;
    } else if (b.origin == b.dest) {
      return EvaluationOrder::GreaterThan;
    } else {
      return EvaluationOrder::NotEqual;
    }
  }

  // else a & b are not adjacent and cannot be compared
  return EvaluationOrder::Unknown;
}

void ConcurrentGPUNeuralNet::sort_connections() {
  if(connections_sorted) {
    return;
  }

  unsigned int max_iterations =
    connections.size()*connections.size()*connections.size()+1;

  bool change_applied = false;
  for(auto i_try=0u; i_try < max_iterations; i_try++) {
    change_applied = false;

    for(auto i=0u; i<connections.size(); i++) {
      for(auto j=i+1; j<connections.size(); j++) {
        Connection& conn1 = connections[i];
        Connection& conn2 = connections[j];

        switch(compare_connections(conn1,conn2)) {
        case EvaluationOrder::GreaterThan:
          if (conn1.set <= conn2.set) {
            conn1.set = conn2.set + 1;
            change_applied = true;
          }
          break;

        case EvaluationOrder::LessThan:
          if(conn2.set <= conn1.set) {
            conn2.set = conn1.set + 1;
            change_applied = true;
          }
          break;

        case EvaluationOrder::NotEqual:
          if(conn1.set == conn2.set) {
            conn2.set = conn1.set + 1;
            change_applied = true;
          }
          break;

        case EvaluationOrder::Unknown:
          break;
        }
      }
    }

    if(!change_applied) {
      break;
    }
  }

  if(change_applied) {
    throw std::runtime_error("Sort Error: change_applied == true on last possible iteration");
  }

  // sort connections based on evaluation set number
  std::sort(connections.begin(),connections.end(),[](Connection a, Connection b){ return a.set < b.set; });
  for (auto const& conn : connections) {
    connection_list.add(conn.origin,conn.dest,conn.weight);
  }
  connections_sorted = true;

  build_action_list();
  connections.clear();
  synchronize();
}

void ConcurrentGPUNeuralNet::ConcurrentGPUNeuralNet::build_action_list() {

  unsigned int num_connection_sets = connections.back().set+1;
  std::vector<unsigned int> connection_set_sizes(num_connection_sets, 0);
  for(auto& conn : connections) {
    connection_set_sizes[conn.set]++;
  }

  std::vector<unsigned int> earliest_zero_out_indices(nodes.size(), 0);
  std::vector<unsigned int> earliest_sigmoid_indices(nodes.size(), 0);

  std::vector<unsigned int> latest_zero_out_indices(nodes.size(), num_connection_sets);
  std::vector<unsigned int> latest_sigmoid_indices(nodes.size(), num_connection_sets);

  std::set<unsigned int> self_recurrent_nodes;

  for(auto& conn : connections) {
    // delay earliest possible zeroing of recurrent connections origins
    // until recurrent connections are applied
    if(conn.type == ConnectionType::Recurrent) {
      earliest_zero_out_indices[conn.origin] = std::max(
        earliest_zero_out_indices[conn.origin],
        conn.set + 1);
    }

    earliest_sigmoid_indices[conn.dest] = std::max(
      earliest_sigmoid_indices[conn.dest],
      conn.set + 1);

    latest_zero_out_indices[conn.dest] = std::min(
      latest_zero_out_indices[conn.dest],
      conn.set);

    if(conn.type == ConnectionType::Normal) {
      latest_sigmoid_indices[conn.origin] = std::min(
        latest_sigmoid_indices[conn.origin],
        conn.set);
    }

    if(conn.origin == conn.dest) {
      self_recurrent_nodes.insert(conn.origin);
    }
  }

  std::vector<unsigned int>& zero_out_indices = earliest_zero_out_indices;
  std::vector<unsigned int>& sigmoid_indices = earliest_sigmoid_indices;


  std::vector<std::vector<unsigned int> > zero_out_sets(num_connection_sets+1);
  std::vector<std::vector<unsigned int> > sigmoid_sets(num_connection_sets+1);

  for(unsigned int i=0; i<nodes.size(); i++) {
    bool is_self_recurrent = self_recurrent_nodes.count(i);
    if(!is_self_recurrent && i >= num_inputs) {
      zero_out_sets[zero_out_indices[i]].push_back(i);
    }
    if(i >= num_inputs) {
      sigmoid_sets[sigmoid_indices[i]].push_back(i);
    }
  }

  auto append_zero_out_set = [&](unsigned int i) {
    auto& zero_out_set = zero_out_sets[i];
    action_list.push_back(zero_out_set.size());
    for(unsigned int j : zero_out_set) {
      action_list.push_back(j);
    }
  };

  auto append_sigmoid_set = [&](unsigned int i) {
    auto& sigmoid_set = sigmoid_sets[i];
    action_list.push_back(sigmoid_set.size());
    for(unsigned int j : sigmoid_set) {
      action_list.push_back(j);
    }
  };



  action_list.clear();
  for(unsigned int i=0; i<num_connection_sets; i++) {
    append_zero_out_set(i);
    append_sigmoid_set(i);
    action_list.push_back(connection_set_sizes[i]);
  }

  append_zero_out_set(num_connection_sets);
  append_sigmoid_set(num_connection_sets);



  // print action list
  // for (auto& item : action_list) {
  //   std::cout << item << " ";
  // } std::cout << std::endl;
}


////////////////////////////////////////////////////////////////////////////

void ConcurrentGPUNeuralNet::add_node(const NodeType& type) {
  switch (type) {
  case NodeType::Bias:
  case NodeType::Input:
    num_inputs++;
    break;
  case NodeType::Output:
    num_outputs++;
    break;
  case NodeType::Hidden:
    break;
  };
  nodes.push_back(0.0);
}

_float_ sigmoid(_float_ val) {
  return 1/(1 + std::exp(-val));
}

void clear_nodes(unsigned int* list, _float_* nodes, unsigned int n) {
  for(auto i=0u; i<n; i++) {
    nodes[list[i]] = 0;
  }
}

void sigmoid_nodes(unsigned int* list, _float_* nodes, unsigned int n) {
  for(auto i=0u; i<n; i++) {
    nodes[list[i]] = sigmoid(nodes[list[i]]);
  }
}

void apply_connections(_float_* node, unsigned int* origin, unsigned int* dest, _float_* weight, unsigned int n) {
  for(auto i=0u; i<n; i++) {
    auto& conn_origin = origin[i];
    auto& conn_dest = dest[i];
    auto& conn_weight = weight[i];
    if(conn_origin == conn_dest) {
      // Special case for self-recurrent nodes
      // Be sure not to zero-out before this step.
      node[conn_origin] *= conn_weight;
    } else {
      node[conn_dest] += conn_weight*node[conn_origin];
    }
  }
}

__device__ _float_ device_sigmoid(_float_ val) {
  return 1/(1 + expf(-val));
}

__global__ void device_clear_nodes(unsigned int* list, _float_* nodes, unsigned int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i<n) {
    nodes[list[i]] = 0;
  }
}

__global__ void device_sigmoid_nodes(unsigned int* list, _float_* nodes, unsigned int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i<n) {
    nodes[list[i]] = device_sigmoid(nodes[list[i]]);
  }
}

__global__ void device_apply_connections(_float_* node, unsigned int* origin, unsigned int* dest, _float_* weight, unsigned int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i<n) {
    auto& conn_origin = origin[i];
    auto& conn_dest = dest[i];
    auto& conn_weight = weight[i];
    if(conn_origin == conn_dest) {
      // Special case for self-recurrent nodes
      // Be sure not to zero-out before this step.
      node[conn_origin] *= conn_weight;
    } else {
      node[conn_dest] += conn_weight*node[conn_origin];
    }
  }
}

std::vector<_float_> ConcurrentGPUNeuralNet::evaluate(std::vector<_float_> inputs) {
  assert(inputs.size() == num_inputs);
  sort_connections();

  // copy inputs in to network
  std::copy(inputs.begin(),inputs.end(),nodes.begin());

  auto i = 0u;
  int how_many_zero_out = action_list[i++];
  clear_nodes(&action_list[i], nodes.data(), how_many_zero_out);
  i += how_many_zero_out;

  int how_many_sigmoid = action_list[i++];
  sigmoid_nodes(&action_list[i], nodes.data(), how_many_sigmoid);
  i += how_many_sigmoid;

  int current_conn = 0;
  while(i<action_list.size()) {
    int how_many_conn = action_list[i++];
    apply_connections(nodes.data(), &connection_list.origin[current_conn], &connection_list.dest[current_conn], &connection_list.weight[current_conn], how_many_conn);
    current_conn += how_many_conn;

    int how_many_zero_out = action_list[i++];
    clear_nodes(&action_list[i], nodes.data(), how_many_zero_out);
    i += how_many_zero_out;

    int how_many_sigmoid = action_list[i++];
    sigmoid_nodes(&action_list[i], nodes.data(), how_many_sigmoid);
    i += how_many_sigmoid;
  }

  return std::vector<_float_> (nodes.begin()+num_inputs,nodes.begin()+num_inputs+num_outputs);
}

std::vector<_float_> ConcurrentGPUNeuralNet::device_evaluate(std::vector<_float_> inputs, unsigned int num_threads) {
  assert(inputs.size() == num_inputs);
  sort_connections();
  unsigned int num_blocks = 0;

  // copy inputs in to network
  //std::copy(inputs.begin(),inputs.end(),nodes.begin());
  cuda_assert(hipMemcpy(node_,inputs.data(),inputs.size()*sizeof(_float_),hipMemcpyHostToDevice));

  auto i = 0u;
  int how_many_zero_out = action_list[i++];
  num_blocks = (how_many_zero_out+num_threads-1)/num_threads;
  if (how_many_zero_out) { device_clear_nodes<<<num_blocks,num_threads>>>(&action_list_[i], node_, how_many_zero_out); }
  i += how_many_zero_out;

  int how_many_sigmoid = action_list[i++];
  num_blocks = (how_many_sigmoid+num_threads-1)/num_threads;
  if (how_many_sigmoid) { device_sigmoid_nodes<<<num_blocks,num_threads>>>(&action_list_[i], node_, how_many_sigmoid); }
  i += how_many_sigmoid;

  int current_conn = 0;
  while(i<action_list.size()) {
    int how_many_conn = action_list[i++];
    num_blocks = (how_many_conn+num_threads-1)/num_threads;
    if (how_many_conn) { device_apply_connections<<<num_blocks,num_threads>>>(node_, &origin_[current_conn], &dest_[current_conn], &weight_[current_conn], how_many_conn); }
    current_conn += how_many_conn;

    int how_many_zero_out = action_list[i++];
    num_blocks = (how_many_zero_out+num_threads-1)/num_threads;
    if (how_many_zero_out) { device_clear_nodes<<<num_blocks,num_threads>>>(&action_list_[i], node_, how_many_zero_out); }
    i += how_many_zero_out;

    int how_many_sigmoid = action_list[i++];
    num_blocks = (how_many_sigmoid+num_threads-1)/num_threads;
    if (how_many_sigmoid) { device_sigmoid_nodes<<<num_blocks,num_threads>>>(&action_list_[i], node_, how_many_sigmoid); }
    i += how_many_sigmoid;
  }
  cuda_assert(hipDeviceSynchronize());
  std::vector<_float_> outputs(num_outputs,0);
  cuda_assert(hipMemcpy(outputs.data(),&node_[num_inputs],num_outputs*sizeof(_float_),hipMemcpyDeviceToHost));

  return outputs;
}

void ConcurrentGPUNeuralNet::add_connection(int origin, int dest, _float_ weight) {
  if(would_make_loop(origin,dest)) {
    connections.emplace_back(origin,dest,ConnectionType::Recurrent,weight);
  } else {
    connections.emplace_back(origin,dest,ConnectionType::Normal,weight);
  }
}

bool ConcurrentGPUNeuralNet::would_make_loop(unsigned int i, unsigned int j) {
  // handle the case of a recurrent connection to itself up front
  if (i == j) { return true; }

  std::vector<bool> reachable(nodes.size(), false);
  reachable[j] = true;

  while (true) {

    bool found_new_node = false;
    for (auto const& conn : connections) {
      // if the origin of this connection is reachable and its
      // desitination is not, then it should be made reachable
      if (reachable[conn.origin] &&
          !reachable[conn.dest] &&
          conn.type == ConnectionType::Normal) {
        // if it is a normal node. if it is the origin of the
        // proposed additional connection (i->j) then it would be
        // a loop
        if (conn.dest == i) {
          // the destination of this reachable connection is
          // the origin of the proposed connection. thus there
          // exists a path from j -> i. So this will be a loop.
          return true;
        }
        else {
          reachable[conn.dest] = true;
          found_new_node = true;
        }
      }
    }
    // no loop detected
    if (!found_new_node) {
      return false;
    }

  }
}

// TODO: implement gpu_smart_pointer to handle GPU memory according to RAII
void ConcurrentGPUNeuralNet::synchronize() {
  cuda_assert(hipMalloc((void**)&node_,nodes.size()*sizeof(_float_)));
  cuda_assert(hipMemcpy(node_,nodes.data(),nodes.size()*sizeof(_float_),hipMemcpyHostToDevice));

  cuda_assert(hipMalloc((void**)&origin_,connection_list.origin.size()*sizeof(unsigned int)));
  cuda_assert(hipMemcpy(origin_,connection_list.origin.data(),connection_list.origin.size()*sizeof(unsigned int),hipMemcpyHostToDevice));

  cuda_assert(hipMalloc((void**)&dest_,connection_list.dest.size()*sizeof(unsigned int)));
  cuda_assert(hipMemcpy(dest_,connection_list.dest.data(),connection_list.dest.size()*sizeof(unsigned int),hipMemcpyHostToDevice));
  
  cuda_assert(hipMalloc((void**)&weight_,connection_list.weight.size()*sizeof(_float_)));
  cuda_assert(hipMemcpy(weight_,connection_list.weight.data(),connection_list.weight.size()*sizeof(_float_),hipMemcpyHostToDevice));

  cuda_assert(hipMalloc((void**)&action_list_,action_list.size()*sizeof(unsigned int)));
  cuda_assert(hipMemcpy(action_list_,action_list.data(),action_list.size()*sizeof(unsigned int),hipMemcpyHostToDevice));
}


void ConcurrentGPUNeuralNet::print_network(std::ostream& os) const {
  std::stringstream ss;
  ss << "Action List: \n\n";

  auto i = 0u;
  int how_many_zero_out = action_list[i++];
  ss << "# Zero out: " << how_many_zero_out << "\n";
  i += how_many_zero_out;

  int how_many_sigmoid = action_list[i++];
  ss << "# Sigmoid: " << how_many_sigmoid << "\n";
  i += how_many_sigmoid;

  int current_conn = 0;
  while(i<action_list.size()) {
    int how_many_conn = action_list[i++];
    ss << "# Connections: " << how_many_conn << "\n";
    current_conn += how_many_conn;

    int how_many_zero_out = action_list[i++];
    ss << "# Zero out: " << how_many_zero_out << "\n";
    i += how_many_zero_out;

    int how_many_sigmoid = action_list[i++];
    ss << "# Sigmoid: " << how_many_sigmoid << "\n";
    i += how_many_sigmoid;
  }
  os << ss.str();
}